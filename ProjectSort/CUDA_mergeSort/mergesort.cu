#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <float.h>

#include <stdlib.h>
#include <time.h>

#include <caliper/cali.h>
#include <caliper/cali-manager.h>
#include <adiak.hpp>

// Assuming array_fill function is defined elsewhere
void array_fill_random(float *array, int size) {
    // Seed the random number generator to get different results each run
    srand(time(NULL));

    for (int i = 0; i < size; i++) {
        // Generate a random float between 0 and 1
        array[i] = (float)rand() / RAND_MAX;
    }
}

// Fill array in sorted order
void array_fill_sorted(float *array, int size) {
    for (int i = 0; i < size; i++) {
        array[i] = (float)i;  // or any other sorted pattern
    }
}

// Fill array in reverse sorted order
void array_fill_reverse_sorted(float *array, int size) {
    for (int i = 0; i < size; i++) {
        array[i] = (float)(size - i - 1);  // reverse order
    }
}

// Fill array with 1% perturbation
void array_fill_perturbed(float *array, int size) {
    array_fill_sorted(array, size);  // Start with a sorted array
    int perturb_count = size / 100;   // 1% of size
    for (int i = 0; i < perturb_count; i++) {
        int index = rand() % size;
        array[index] = (float)rand() / RAND_MAX;  // Random perturbation
    }
}

bool is_sorted(float *array, int size) {
    for (int i = 0; i < size - 1; i++) {
        if (array[i] > array[i + 1]) {
            return false; // Not sorted
        }
    }
    return true;
}

void adiak_stuff() {
    adiak::init(NULL);
    adiak::launchdate();    // launch date of the job
    adiak::libraries();     // Libraries used
    adiak::cmdline();       // Command line used to launch the job
    adiak::clustername();   // Name of the cluster
    adiak::value("Algorithm", "MergeSort"); // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
    adiak::value("ProgrammingModel", "CUDA"); // e.g., "MPI", "CUDA", "MPIwithCUDA"
    adiak::value("Datatype", "float"); // The datatype of input elements (e.g., double, int, float)
    adiak::value("SizeOfDatatype", sizeof(float)); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
    // adiak::value("InputSize", array_size); // The number of elements in input dataset (1000)
    adiak::value("InputType", "Random"); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
    // adiak::value("num_procs", size); // The number of processors (MPI ranks)
    adiak::value("group_num", 2); // The number of your group (integer, e.g., 1, 10)
    adiak::value("implementation_source", "AI"); // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").
}


int THREADS;
int BLOCKS;
int NUM_VALS;

__global__ void mergeKernel(float *dev_values, int size, int mid, int upper) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= upper) return; // Check if the thread is within the range

    // Calculate starting points of the two halves
    int start1 = idx > mid ? mid : idx;
    int start2 = idx > mid ? idx : mid;
    int end1 = mid, end2 = upper;

    // Calculate the size needed for the temporary array in shared memory
    int tempSize = upper - start1;
    extern __shared__ float temp[];

    // Merge the two halves into temp
    int i = start1, j = start2, k = 0;
    while (i < end1 && j < end2) {
        if (dev_values[i] < dev_values[j]) {
            temp[k++] = dev_values[i++];
        } else {
            temp[k++] = dev_values[j++];
        }
    }

    // Copy the remaining elements of the first half, if any
    while (i < end1) {
        temp[k++] = dev_values[i++];
    }

    // Copy the remaining elements of the second half, if any
    while (j < end2) {
        temp[k++] = dev_values[j++];
    }

    // Copy back the merged elements to the original array
    k = 0;
    for (i = start1; i < upper; i++) {
        dev_values[i] = temp[k++];
    }
}


void mergeSort(float *dev_values, int size) {
    dim3 threads(THREADS, 1);
    dim3 blocks(BLOCKS, 1);

    // Calculate shared memory size per block
    int sharedMemSize = size * sizeof(float);
    
    CALI_MARK_BEGIN("comp");
    CALI_MARK_BEGIN("comp_large");
    for (int width = 1; width < size; width *= 2) {
        for (int i = 0; i < size; i = i + 2 * width) {
            mergeKernel<<<blocks, threads, sharedMemSize>>>(dev_values, size, i, min(i + 2 * width, size));
            // Check for errors after kernel launch
            hipError_t err = hipGetLastError();
            if (err != hipSuccess) {
                fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
                exit(EXIT_FAILURE);
            }
        }
        hipDeviceSynchronize();
    }
    CALI_MARK_END("comp_large");
    CALI_MARK_END("comp");
}

int main(int argc, char *argv[]) {
    // Parse command line arguments
    if (argc < 4) {
        fprintf(stderr, "Usage: %s <number of threads> <number of values> <input type>\n", argv[0]);
        exit(EXIT_FAILURE);
    }

    THREADS = atoi(argv[1]);
    NUM_VALS = atoi(argv[2]);
    BLOCKS = (NUM_VALS + THREADS - 1) / THREADS;
    char* inputType = argv[3]; // New argument for input type


    printf("Number of threads: %d\n", THREADS);
    printf("Number of values: %d\n", NUM_VALS);
    printf("Number of blocks: %d\n", BLOCKS);
    printf("Input type: %s\n", inputType);

    float total = 0.0f, minTime = FLT_MAX, maxTime = 0.0f;
    const int numRuns = 10;
    float times[numRuns];

    // Allocate host array
    float *values = (float*) malloc(NUM_VALS * sizeof(float));
    CALI_MARK_BEGIN("main");

    // Allocate device memory
    float *dev_values;
    hipMalloc((void**) &dev_values, NUM_VALS * sizeof(float));

    for (int run = 0; run < numRuns; ++run) {

        // Fill the array based on input type
        if (strcmp(inputType, "Sorted") == 0) {
            array_fill_sorted(values, NUM_VALS);
        }
        else if (strcmp(inputType, "ReverseSorted") == 0) {
            array_fill_reverse_sorted(values, NUM_VALS);
        }
        else if (strcmp(inputType, "Random") == 0) {
            array_fill_random(values, NUM_VALS);
        }
        else if (strcmp(inputType, "1%Perturbed") == 0) {
            array_fill_perturbed(values, NUM_VALS);
        }
        else {
            fprintf(stderr, "Invalid input type!\n");
            exit(EXIT_FAILURE);
        }

        // Copy data from host to device
        hipMemcpy(dev_values, values, NUM_VALS * sizeof(float), hipMemcpyHostToDevice);

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        mergeSort(dev_values, NUM_VALS);
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&times[run], start, stop);
        total += times[run];
        if (times[run] < minTime) minTime = times[run];
        if (times[run] > maxTime) maxTime = times[run];

        hipEventDestroy(start);
        hipEventDestroy(stop);

        // Copy sorted data back to host and verify
        hipMemcpy(values, dev_values, NUM_VALS * sizeof(float), hipMemcpyDeviceToHost);
        if (!is_sorted(values, NUM_VALS)) {
            fprintf(stderr, "Array is not correctly sorted!\n");
            exit(EXIT_FAILURE);
        }
    }

    float average = total / numRuns;

    CALI_MARK_END("main");

    // Adiak value updates
    adiak::init(NULL);
    adiak::launchdate();    // launch date of the job
    adiak::libraries();     // Libraries used
    adiak::cmdline();       // Command line used to launch the job
    adiak::clustername();   // Name of the cluster
    adiak::value("Algorithm", "MergeSort"); // The name of the algorithm you are using (e.g., "MergeSort", "BitonicSort")
    adiak::value("ProgrammingModel", "CUDA"); // e.g., "MPI", "CUDA", "MPIwithCUDA"
    adiak::value("Datatype", "float"); // The datatype of input elements (e.g., double, int, float)
    adiak::value("SizeOfDatatype", sizeof(float)); // sizeof(datatype) of input elements in bytes (e.g., 1, 2, 4)
    // adiak::value("InputSize", array_size); // The number of elements in input dataset (1000)
    adiak::value("InputType", "Random"); // For sorting, this would be "Sorted", "ReverseSorted", "Random", "1%perturbed"
    // adiak::value("num_procs", size); // The number of processors (MPI ranks)
    adiak::value("group_num", 2); // The number of your group (integer, e.g., 1, 10)
    adiak::value("implementation_source", "AI"); // Where you got the source code of your algorithm; choices: ("Online", "AI", "Handwritten").

    // Print timing statistics
    printf("Average GPU Time: %f ms\n", average);
    printf("Minimum GPU Time: %f ms\n", minTime);
    printf("Maximum GPU Time: %f ms\n", maxTime);
    printf("Total GPU Time: %f ms\n", total);

    // Clean up
    hipFree(dev_values);
    free(values);

    return 0;
}
